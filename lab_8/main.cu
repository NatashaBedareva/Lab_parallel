#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <cmath>
#include <string>
#include <iomanip>
#include <tuple>
#include <chrono>
#include <cstdlib>
#include <boost/program_options.hpp>

namespace po = boost::program_options;

int NX = 20;      
int NY = 20;     
double TAU = -0.01;  
double EPS = 0.01;   
int MAX_ITER = 1000;
const std::string OUT_FILE = "result.dat";  

using namespace std::chrono;

const std::tuple<int, int, double> sources[] = {
    {NY/2, NX/3, 10},      
    {NY*2/3, NX*2/3, -25}, 
    {5, 10, 15},           
    {15, 25, -10},
    {10, 5, 20}
};
const int NUM_SOURCES = sizeof(sources) / sizeof(sources[0]);

double get_a(int row, int col) {
    if (row == col) return -4.0;
    if ((row + 1 == col && col % NX != 0) || 
        (row - 1 == col && row % NX != 0) ||  
        (row + NX == col && col < (NX * NY)) || 
        (row - NX == col && row >= NX))       
        return 1.0;
    return 0.0;
}

double get_b(int idx) {
    int i = idx / NX;
    int j = idx % NX;
    
    for (int k = 0; k < NUM_SOURCES; k++) {
        if (std::get<0>(sources[k]) == i && std::get<1>(sources[k]) == j) {
            return std::get<2>(sources[k]);
        }
    }
    return 0.0;
}

void init_matrix(double* A, int SIZE) {
    for (int i = 0; i < SIZE; ++i) {
        for (int j = 0; j < SIZE; ++j) {
            A[i * SIZE + j] = get_a(i, j);
        }
    }
}

void init_b(double* b, int SIZE) {
    for (int i = 0; i < SIZE; ++i) {
        b[i] = get_b(i);
    }
}

double norm(const double* vec, int size) 
{
    double result = 0.0;
    for (int i = 0; i < size; ++i) {
        result += vec[i] * vec[i];
    }
    return std::sqrt(result);
}

__global__ void norm_kernel(double* result, const double* vec, int size) {
    extern __shared__ double shared_mem[];
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + tid;
    
    shared_mem[tid] = (i < size) ? vec[i] * vec[i] : 0.0;
    __syncthreads();
    
    for (int s = blockDim.x/2; s > 0; s >>= 1) {
        if (tid < s) {
            shared_mem[tid] += shared_mem[tid + s];
        }
        __syncthreads();
    }
    
    if (tid == 0) {
        atomicAdd(result, shared_mem[0]);
    }
}

//Ax - b
__global__ void matrix_vector_mult_sub_kernel(double* res, const double* mat, const double* vec, const double* y, int SIZE) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < SIZE) {
        double sum = -y[i];
        for (int j = 0; j < SIZE; ++j) {
            sum += mat[i * SIZE + j] * vec[j];
        }
        res[i] = sum;
    }
}


__global__ void update_solution_kernel(double* x, const double* delta, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        x[i] -= -0.01f * delta[i];
    }
}

void solve_simple_iteration(double* A,
                           double* x,
                           const double* b,
                           int SIZE) {
    double norm_b = norm(b, SIZE);
    double h_res_norm = 0.0;
    int iter = 0;

    double *d_mat, *d_vec, *d_b, *d_res, *d_res_norm;
    
    hipMalloc(&d_mat, SIZE * SIZE * sizeof(double));
    hipMalloc(&d_vec, SIZE * sizeof(double));
    hipMalloc(&d_b, SIZE * sizeof(double));
    hipMalloc(&d_res, SIZE * sizeof(double));
    hipMalloc(&d_res_norm, sizeof(double));

    hipMemcpy(d_mat, A, SIZE * SIZE * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, SIZE * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_vec, x, SIZE * sizeof(double), hipMemcpyHostToDevice);
    
    int threadsPerBlock = 256;
    int blocksPerGrid = (SIZE + threadsPerBlock - 1) / threadsPerBlock;


    auto start = high_resolution_clock::now();



    do {
        
        h_res_norm = 0.0;
        hipMemcpy(d_res_norm, &h_res_norm, sizeof(double), hipMemcpyHostToDevice);
        
        matrix_vector_mult_sub_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_res, d_mat, d_vec, d_b, SIZE);
        hipDeviceSynchronize();
        
        norm_kernel<<<blocksPerGrid, threadsPerBlock, threadsPerBlock*sizeof(double)>>>(d_res_norm, d_res, SIZE);
        hipDeviceSynchronize();

        update_solution_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_vec, d_res, SIZE);
        hipDeviceSynchronize();

        hipMemcpy(&h_res_norm, d_res_norm, sizeof(double), hipMemcpyDeviceToHost);
        h_res_norm = sqrt(h_res_norm);

        std::cout << "Iteration " << std::setw(4) << ++iter 
                  << ": residual = " << std::scientific << std::setprecision(6) 
                  << h_res_norm/norm_b << " (target < " << EPS << ")\r";
        std::cout.flush();
        
        if (iter >= MAX_ITER) {
            std::cout << "\nMaximum iterations (" << MAX_ITER << ") reached\n";
            break;
        }
    } while (h_res_norm/norm_b >= EPS);

    hipMemcpy(x, d_vec, SIZE * sizeof(double), hipMemcpyDeviceToHost);

    if (iter < MAX_ITER) {
        std::cout << "\nConverged after " << iter << " iterations\n";
    }

    auto stop = high_resolution_clock::now();
    auto duration = duration_cast<microseconds>(stop - start);
    printf("Time run = %.5f s\n",float(duration.count())/1000000);
    
    hipFree(d_mat);
    hipFree(d_vec);
    hipFree(d_b);
    hipFree(d_res);
    hipFree(d_res_norm);
}


void save_results(const double* x, int size) {
    std::ofstream out(OUT_FILE, std::ios::binary);
    out.write(reinterpret_cast<const char*>(x), size * sizeof(double));
}

//./main--nx 10 --ny 10 --tau -0.005 --eps 0.001 --max-iter 5000

int main(int argc, char* argv[]) {
   
    po::options_description desc("Allowed options");
    desc.add_options()
        ("help,h", "produce help message")
        ("nx", po::value<int>(), "set grid size in X direction")
        ("ny", po::value<int>(), "set grid size in Y direction")
        ("tau", po::value<double>(), "set iteration parameter tau")
        ("eps", po::value<double>(), "set precision epsilon")
        ("max-iter", po::value<int>(), "set maximum number of iterations")
    ;


    po::variables_map vm;
    po::store(po::parse_command_line(argc, argv, desc), vm);
    po::notify(vm);

    if (vm.count("help")) {
        std::cout << desc << "\n";
        return 1;
    }

    if (vm.count("nx")) {
        NX = vm["nx"].as<int>();
    }
    if (vm.count("ny")) {
        NY = vm["ny"].as<int>();
    }
    if (vm.count("tau")) {
        TAU = vm["tau"].as<double>();
    }
    if (vm.count("eps")) {
        EPS = vm["eps"].as<double>();
    }
    if (vm.count("max-iter")) {
        MAX_ITER = vm["max-iter"].as<int>();
    }

    const int SIZE = NX * NY;
     

    double* A;
    double* b;
    double* x;

    A = (double*)malloc(SIZE * SIZE * sizeof(double));
    b = (double*)malloc(SIZE * sizeof(double));
    x = (double*)malloc(SIZE * sizeof(double));

    for (int i = 0; i < SIZE; ++i) {
        x[i] = 0.0;
    }

    
    std::cout << "Solving heat distribution on " << NY << "x" << NX << " grid\n";
    std::cout << "With " << NUM_SOURCES << " heat sources/sinks\n";
    std::cout << "Parameters: tau=" << TAU << ", eps=" << EPS << ", max_iter=" << MAX_ITER << "\n";

    init_matrix(A, SIZE);
    init_b(b, SIZE);


    solve_simple_iteration(A, x, b, SIZE);
    

    save_results(x, SIZE);

    free(A);
    free(b);
    free(x);

    return 0;
}